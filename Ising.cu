#include "hip/hip_runtime.h"
//-code=sm_20;
//Compile: nvcc Ising.cu -o Isingcuda
//Run: ./Isingcuda
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdlib.h>

/*
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
*/

__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed,threadIdx.x, 51,&states[threadIdx.x]);
}

__device__ int CalcE(int N,int arr[])
{        
       	int i,j;
	int M = 0;
	int E = 0;
	for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			M += arr[j+N*i];
			if(i<N-1){
				E+= -arr[j+N*i] * arr[j+1+N*i];
			}
			else if(i ==N-1){
				E+= -arr[j+N*i] * arr[j];
			}
			if(j<N-1){
				E+= -arr[j+N*i] * arr[j+1+N*i];
			}
			else if(j ==N-1){
				E+= -arr[j+N*i] * arr[j];
			}
		}
	}
	return E;
}

__device__ int Rel_CalcE(int N,int arr[] , int x, int y)
{
	//Calculate energy of nearest neighbors
	int E = 0;
	int i = x;
	int j = y;
	if(j>0 && j<N-1){
		E+= -arr[j+N*i] * arr[j+N*i+1]-arr[j+N*i] * arr[j+N*i-1];
	}
	else if(j ==N-1){
		E+= -arr[j+N*i] * arr[N*i]-arr[j+N*i] * arr[j+N*i-1];
	}
	else if(j ==0){
		E+= -arr[j+N*i] * arr[j+N*i+1]-arr[j+N*i] * arr[N*i+N-1];
	}
	if(i>0 && i<N-1){
		E+= -arr[j+N*i] * arr[j+N*(i+1)]-arr[j+N*i] * arr[j+N*(i-1)];
	}
	else if(i ==N-1){
		E+= -arr[j+N*i] * arr[i]-arr[j+N*i] * arr[j+N*(i-1)];
	}
	else if(i ==0){
		E+= -arr[j+N*i] * arr[j+N*(i+1)]-arr[j+N*i] * arr[j+N*(N-1)];
	}
	return E;

}

__device__ int flip(int N,int arr[],hiprandState_t* states)
{
	float p,num;
	float temp = threadIdx.x/10.;
	int i,j,Enew,E0,E= 0;
	for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			E0 = Rel_CalcE(N,arr,i,j);
			Enew = E0*(-1);
			if(Enew<= E0){
				arr[j+N*i] *= -1;
			}

			else{
				p = exp(-(Enew-E0)/temp);
				num = (hiprand(&states[threadIdx.x])% 101)/100.;
				if(p>=num){
					arr[j+N*i] *= -1;
				}			
			}				
		}
	}
	E = CalcE(N,arr);
	return E;
}

__global__ void Ising(int *arr, float *Earr,int *N,hiprandState_t* states)
{
  int itt = 100;
  int k = 0;
  int x = threadIdx.x;
  int y =(int ) *N;
  Earr[x]=0;

  for (k = 0; k<itt; k++){ 
  	Earr[x] += flip(y,arr);
  }
  Earr[x] = Earr[x]/itt;


}

int main()
{
    int i,j,num,k;
    int N;
    FILE *fd;
    FILE *ft;
    srand (time(NULL));
    hiprandState_t* states;
    fd=fopen("dataCuda.txt", "w");
    ft=fopen("timeCuda.txt", "w");
    for(N=10; N<52; N=N+2){

 	 /* allocate space on the GPU for the random states */
 	 hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  	 /* invoke the GPU to initialize all of the random states */
 	 init<<<1, 51>>>(time(0), states);

   	 size_t arrsize = N*N * sizeof(int);
   	 size_t Esize = 51 * sizeof(double);
	 size_t Nsize = sizeof(int);
   	 clock_t start, end;
   	 double cpu_time_used;
   	 start = clock();

   	 int* h_Arr = (int*)malloc(arrsize);
   	 float* h_E = (float*)malloc(51 * sizeof(float));


   	 // Allocate vectors in device memory
   	 int *d_Arr, *d_N;	
	 float *d_E;
   	 hipMalloc(&d_Arr, arrsize);
   	 hipMalloc(&d_E, Esize);
	 hipMalloc(&d_N, Nsize);
	 //cudaCheckErrors("cudamalloc fail");

   	 for (i = 0; i<k; i++){
		for (j = 0; j<k; j++){
			num = rand() % 101;
			if(num<50){
				h_Arr[j+i*k] = 1;
			}
			else{
				h_Arr[j+i*k] = -1;
			}
			//printf("%d", h_Arr[j+i*k]);
			}

		}
   	 // Copy vectors from host memory to device memory
    	 hipMemcpy(d_Arr, &h_Arr, arrsize, hipMemcpyHostToDevice);
   	 hipMemcpy(d_E, h_E, Esize, hipMemcpyHostToDevice);
	hipMemcpy(d_N, &N, Nsize, hipMemcpyHostToDevice);
	//cudaCheckErrors("cuda memcpy fail");

    	// Invoke kernel
   	 dim3 dimBlock( 51 );
   	 dim3 dimGrid( 1 );
    	Ising<<<dimGrid, dimBlock>>>(d_Arr, d_E, d_N, states);
   	 // Copy result from device memory to host memory
   	 hipMemcpy(h_E, d_E, Esize, hipMemcpyDeviceToHost);
    	hipMemcpy(h_Arr, d_Arr, arrsize, hipMemcpyDeviceToHost);
	//cudaCheckErrors("cudamemcpy or cuda kernel fail");
    	// Free device memory
   	hipFree(d_Arr);
    	hipFree(d_E);
	hipFree(d_N);
	hipFree(states);
    	end = clock();
    	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	fprintf(fd, "%f\n", float(h_E[i]/(N*N*100.)));
	fprintf(ft, "%f\n", cpu_time_used);	
	}
	fclose(fd);
	fclose(ft);
    return EXIT_SUCCESS;
}
